#include "hip/hip_runtime.h"
// cuda can't handle gcc 4.7 includes...
// https://bugs.archlinux.org/task/29359
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include "librta/basic_types.h"
#include "bbvh-cuda-node.h"

#include <librta/cuda-kernels.h>
#include <librta/cuda-vec.h>
#include <librta/intersect.h>

#include <iostream>
#include <stdio.h>
using namespace std;

namespace rta {
	namespace cuda {
			
		#define ray_x (blockIdx.x * blockDim.x + threadIdx.x)
		#define ray_y (blockIdx.y * blockDim.y + threadIdx.y)

		namespace k {
			__global__ void trace_dis(cuda::simple_triangle *triangles, int n, bbvh::node<cuda::simple_aabb> *nodes, 
			                          vec3f *ray_orig, vec3f *ray_dir, float *max_t, 
			                          int w, int h, triangle_intersection<simple_triangle> *intersections) {
				if (ray_x < w && ray_y < h) {
					uint tid = ray_y*w+ray_x;
					uint32_t stack[32];
					stack[0] = 0;
					int sp = 0;
					vec3f orig = (ray_orig)[tid];
					vec3f dir = (ray_dir)[tid];
					float t_max = max_t[tid];
					simple_aabb box;
					float dist;
					triangle_intersection<simple_triangle> closest = intersections[tid];
					closest.t = FLT_MAX;
					while (sp >= 0) {
						uint node = stack[sp--];
						bbvh::node<simple_aabb> curr = nodes[node];
						if (curr.inner()) {
							if (intersect_aabb(curr.box, &orig, &dir, dist))
								if (dist < closest.t && dist <= t_max) {
									stack[++sp] = curr.right();
									stack[++sp] = curr.left();
								}
						}
						else {
							uint elems = curr.elems();
							uint offset = curr.tris();
							for (int i = 0; i < elems; ++i) {
								triangle_intersection<simple_triangle> is(offset+i);
								if (intersect_tri_opt(triangles[offset+i], &orig, &dir, is)) {
									if (is.t < closest.t && is.t <= t_max)
										closest = is;
								}
							}
						}
					}
					intersections[tid] = closest;
				}
			}
		}
		
		#define checked_cuda(ans) { gpu_assert((ans), __FILE__, __LINE__); }
		inline void gpu_assert(hipError_t code, char *file, int line, bool abort=true) {
			if (code != hipSuccess) {
				fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
				if (abort) exit(code);
			}
		}


		void trace_dis(simple_triangle *triangles, int n, bbvh::node<simple_aabb> *nodes, vec3f *ray_orig, vec3f *ray_dir, float *max_t, 
		               int w, int h, triangle_intersection<simple_triangle> *is) {
			checked_cuda(hipPeekAtLastError());
			dim3 threads(16, 16);
			dim3 blocks = block_configuration_2d(w, h, threads);
			k::trace_dis<<<blocks, threads>>>(triangles, n, nodes, ray_orig, ray_dir, max_t, w, h, is);
			checked_cuda(hipPeekAtLastError());
			checked_cuda(hipDeviceSynchronize());
		}

	}
}
